//Copyright ETH Zurich, IWF

//This file is part of mfree_iwf-ul_cut_gpu.

//mfree_iwf is free software: you can redistribute it and/or modify
//it under the terms of the GNU General Public License as published by
//the Free Software Foundation, either version 3 of the License, or
//(at your option) any later version.

//mfree_iwf-ul_cut_gpu is distributed in the hope that it will be useful,
//but WITHOUT ANY WARRANTY; without even the implied warranty of
//MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//GNU General Public License for more details.

//You should have received a copy of the GNU General Public License
//along with mfree_iwf-ul_cut_gpu.  If not, see <http://www.gnu.org/licenses/>.

#include "particle_gpu.h"

particle_gpu::particle_gpu(unsigned int N) {
	hipMalloc((void **) &pos, sizeof(float2_t)*N);
	hipMalloc((void **) &vel, sizeof(float2_t)*N);
#ifdef TVF
	hipMalloc((void **) &vel_adv, sizeof(float2_t)*N);
#endif

	hipMalloc((void **) &h,   sizeof(float_t)*N);
	hipMalloc((void **) &rho, sizeof(float_t)*N);
	hipMalloc((void **) &p,   sizeof(float_t)*N);

	hipMalloc((void **) &S,   sizeof(float4_t)*N);
	hipMalloc((void **) &R,   sizeof(float4_t)*N);
	hipMalloc((void **) &fc,  sizeof(float2_t)*N);
	hipMalloc((void **) &ft,  sizeof(float2_t)*N);
	hipMalloc((void **) &n,   sizeof(float2_t)*N);

	hipMalloc((void**) &fixed, sizeof(float_t)*N);
	hipMalloc((void**) &blanked, sizeof(float_t)*N);
	hipMalloc((void**) &tool_particle, sizeof(float_t)*N);

	hipMalloc((void**) &eps_pl, sizeof(float_t)*N);
	hipMalloc((void**) &eps_pl_dot, sizeof(float_t)*N);
	hipMalloc((void**) &T, sizeof(float_t)*N);

	hipMalloc((void **) &pos_t, sizeof(float2_t)*N);
	hipMalloc((void **) &vel_t, sizeof(float2_t)*N);
#ifdef TVF
	hipMalloc((void **) &vel_adv_t, sizeof(float2_t)*N);
#endif
	hipMalloc((void **) &rho_t, sizeof(float_t)*N);
	hipMalloc((void **) &S_t,   sizeof(float4_t)*N);
	hipMalloc((void **) &T_t,   sizeof(float_t)*N);

	hipMalloc((void **) &v_der, sizeof(float4_t)*N);
	hipMalloc((void **) &S_der, sizeof(float4_t)*N);

	hipMalloc((void **) &idx, sizeof(int)*N);
	hipMalloc((void **) &hash, sizeof(int)*N);

	thrust::device_ptr<int> t_idx(this->idx);
	thrust::sequence(t_idx, t_idx+N);

	hipMemset(pos, 0, sizeof(float2_t)*N);
	hipMemset(vel,0, sizeof(float2_t)*N);

	hipMemset(h,0,   sizeof(float_t)*N);
	hipMemset(rho,0, sizeof(float_t)*N);
	hipMemset(p,0,   sizeof(float_t)*N);

	hipMemset(S,0,   sizeof(float4_t)*N);
	hipMemset(R,0,   sizeof(float4_t)*N);
	hipMemset(fc,0,  sizeof(float2_t)*N);
	hipMemset(ft,0,  sizeof(float2_t)*N);
	hipMemset(n,0,   sizeof(float2_t)*N);

	hipMemset(fixed,0,  sizeof(float_t)*N);
	hipMemset(blanked,0,  sizeof(float_t)*N);
	hipMemset(fixed,0,  sizeof(float_t)*N);
	hipMemset(tool_particle, 0, sizeof(float_t)*N);

	hipMemset(eps_pl, 0, sizeof(float_t)*N);
	hipMemset(eps_pl_dot, 0, sizeof(float_t)*N);
	hipMemset(T, 0, sizeof(float_t)*N);

	hipMemset(pos_t,0, sizeof(float2_t)*N);
	hipMemset(vel_t,0, sizeof(float2_t)*N);
	hipMemset(rho_t,0, sizeof(float_t)*N);
	hipMemset(S_t,0,   sizeof(float4_t)*N);
	hipMemset(T_t,0,   sizeof(float_t)*N);

	hipMemset(v_der, 0, sizeof(float4_t)*N);
	hipMemset(S_der, 0, sizeof(float4_t)*N);

	this->N = N;
}

particle_gpu::particle_gpu(float2_t *pos, float2_t *vel_init, float_t *rho, float_t *h, unsigned int N) : particle_gpu(N) {
	hipMemcpy(this->pos, pos,      sizeof(float2_t)*N, hipMemcpyHostToDevice);
	hipMemcpy(this->vel, vel_init, sizeof(float2_t)*N, hipMemcpyHostToDevice);
	hipMemcpy(this->rho, rho,      sizeof(float_t)*N, hipMemcpyHostToDevice);
	hipMemcpy(this->h,   h,        sizeof(float_t)*N, hipMemcpyHostToDevice);
}

particle_gpu::particle_gpu(float2_t *pos, float2_t *vel_init, float_t *rho, float_t *h, float_t *fixed, unsigned int N)
: particle_gpu(pos, vel_init, rho, h, N) {
	hipMemcpy(this->fixed, fixed, sizeof(float_t)*N, hipMemcpyHostToDevice);
}

particle_gpu::particle_gpu(float2_t *pos, float2_t *vel_init, float_t *rho, float_t *T_init, float_t *h, float_t *fixed, unsigned int N)
: particle_gpu(pos, vel_init, rho, h, fixed, N) {
	hipMemcpy(this->T, T_init, sizeof(float_t)*N, hipMemcpyHostToDevice);
}

particle_gpu::particle_gpu(float2_t *pos, float2_t *vel_init, float_t *rho, float_t *T_init, float_t *h, float_t *fixed, float_t * tool_p, unsigned int N)
: particle_gpu(pos, vel_init, rho, T_init, h, fixed, N) {
	hipMemcpy(this->tool_particle, tool_p, sizeof(float_t)*N, hipMemcpyHostToDevice);
}

particle_gpu::particle_gpu(float2_t *pos, float2_t *vel_init, float_t *rho, float_t *h, float4_t *S, unsigned int N) : particle_gpu(N) {
	hipMemcpy(this->pos, pos,      sizeof(float2_t)*N, hipMemcpyHostToDevice);
	hipMemcpy(this->vel, vel_init, sizeof(float2_t)*N, hipMemcpyHostToDevice);
	hipMemcpy(this->rho, rho,      sizeof(float_t)*N, hipMemcpyHostToDevice);
	hipMemcpy(this->h,   h,        sizeof(float_t)*N, hipMemcpyHostToDevice);
	hipMemcpy(this->S,   S,        sizeof(float4_t)*N, hipMemcpyHostToDevice);
}
